#include "hip/hip_runtime.h"
extern "C"

/*Vector additional ketnel*/
__global__ void add(int N, float *vectorA, float *vectorB, float *resultC)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
	/*Limit of operation*/
    if (i<N)
    {
        resultC[idx] = vectorA[idx] + vectorB[idx];
    }
}
